#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <chrono>
#include <cmath>
#include <fstream>
#include <stdexcept>

#include "common.hpp"
#include "obstacle.hpp"

// CUDA用のデバイスメモリポインタ
double* d_rewards;
double* d_values;
double* d_new_values;

Action* d_actions;

// CUDAカーネル関数
__global__ void calculate_value_kernel(double* d_rewards, double* d_values,
                                       double* d_new_values, Action* d_actions,
                                       int size, int theta_size, double gamma,
                                       int num_actions) {
  // スレッドのインデックスを計算
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int theta = blockIdx.z * blockDim.z + threadIdx.z;

  // グリッドの範囲内かチェック
  if (i < size && j < size && theta < theta_size) {
    double max_value = -1e9;
    // 各アクションに対して価値を計算
    for (int k = 0; k < num_actions; ++k) {
      int di = d_actions[k].di;
      int dj = d_actions[k].dj;
      int dtheta = d_actions[k].dtheta;
      int ni = i + di;
      int nj = j + dj;
      int ntheta = (theta + dtheta + theta_size) % theta_size;
      if (ni >= 0 && ni < size && nj >= 0 && nj < size) {
        double cost_multiplier =
            (abs(di) == 1 && abs(dj) == 1) ? sqrt(2.0) : 1.0;
        double new_value =
            d_rewards[ni * size + nj] * cost_multiplier +
            gamma * d_values[(ni * size + nj) * theta_size + ntheta];
        if (new_value > max_value) {
          max_value = new_value;
        }
      }
    }
    d_new_values[(i * size + j) * theta_size + theta] = max_value;
  }
}

// CUDAメモリ配列の初期化関数
void initialize_cuda_memory(const Matrix2D& rewards, const Matrix3D& values,
                            const std::vector<Action>& actions, int size,
                            int theta_size) {
  int num_elements = size * size * theta_size;
  int reward_elements = size * size;

  hipMalloc(&d_rewards, reward_elements * sizeof(double));
  hipMalloc(&d_values, num_elements * sizeof(double));
  hipMalloc(&d_new_values, num_elements * sizeof(double));
  hipMalloc(&d_actions, actions.size() * sizeof(Action));

  // ホストメモリ側要素保存場所を作成
  std::vector<double> h_rewards(reward_elements);
  std::vector<double> h_values(num_elements);

  // CUDAで使用する1次元のデータ構造に変換
  for (int i = 0; i < size; ++i) {
    for (int j = 0; j < size; ++j) {
      h_rewards[i * size + j] = rewards[i][j];
      for (int theta = 0; theta < theta_size; ++theta) {
        h_values[(i * size + j) * theta_size + theta] = values[i][j][theta];
      }
    }
  }

  // デバイスメモリにコピー
  hipMemcpy(d_rewards, h_rewards.data(), reward_elements * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(d_values, h_values.data(), num_elements * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(d_actions, actions.data(), actions.size() * sizeof(Action),
             hipMemcpyHostToDevice);
}

// 価値反復を実行する関数
void execute_value_iteration(int size, int theta_size, double gamma,
                             int max_iterations, double threshold,
                             const std::vector<Action>& actions) {
  // グリッドのサイズを設定
  dim3 blockDim(8, 8, 8);
  dim3 gridDim((size + blockDim.x - 1) / blockDim.x,
               (size + blockDim.y - 1) / blockDim.y,
               (theta_size + blockDim.z - 1) / blockDim.z);

  std::vector<double> h_values(size * size * theta_size);
  std::vector<double> h_new_values(size * size * theta_size);

  for (int iter = 0; iter < max_iterations; ++iter) {
    // カーネル関数の呼び出し
    calculate_value_kernel<<<gridDim, blockDim>>>(
        d_rewards, d_values, d_new_values, d_actions, size, theta_size, gamma,
        actions.size());

    // 同期処理
    hipDeviceSynchronize();

    // 計算結果をホストメモリにコピー
    hipMemcpy(h_values.data(), d_values,
               size * size * theta_size * sizeof(double),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_new_values.data(), d_new_values,
               size * size * theta_size * sizeof(double),
               hipMemcpyDeviceToHost);

    // 収束判定
    double max_delta = 0.0;
    for (int i = 0; i < size; ++i) {
      for (int j = 0; j < size; ++j) {
        for (int theta = 0; theta < theta_size; ++theta) {
          int idx = (i * size + j) * theta_size + theta;
          max_delta =
              std::max(max_delta, std::abs(h_values[idx] - h_new_values[idx]));
        }
      }
    }

    if (max_delta < threshold) {
      std::cout << "Converged after " << iter + 1
                << " iterations with max delta: " << max_delta << std::endl;
      break;
    }

    hipMemcpy(d_values, h_new_values.data(),
               size * size * theta_size * sizeof(double),
               hipMemcpyHostToDevice);
  }
}

// GPUの情報を表示する関数
void print_gpu_info() {
  int device_count;
  hipGetDeviceCount(&device_count);

  if (device_count == 0) {
    std::cout << "No CUDA-compatible GPU detected." << std::endl;
    return;
  }

  for (int i = 0; i < device_count; ++i) {
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, i);

    std::cout << "GPU " << i << ": " << device_prop.name << std::endl;
    std::cout << "CUDA Cores: " << device_prop.multiProcessorCount * 128
              << std::endl;
    std::cout << "Clock Rate: " << device_prop.clockRate / 1000 << " MHz"
              << std::endl;
    std::cout << "Global Memory: " << device_prop.totalGlobalMem / (1 << 20)
              << " MB" << std::endl;
  }
}

// 結果を保存する関数
void save_results(const std::string& filename, int size, int theta_size) {
  Matrix3D values(size, std::vector<std::vector<double>>(
                            size, std::vector<double>(theta_size)));

  std::vector<double> h_values(size * size * theta_size);
  hipMemcpy(h_values.data(), d_values,
             size * size * theta_size * sizeof(double), hipMemcpyDeviceToHost);

  for (int i = 0; i < size; ++i) {
    for (int j = 0; j < size; ++j) {
      for (int theta = 0; theta < theta_size; ++theta) {
        values[i][j][theta] = h_values[(i * size + j) * theta_size + theta];
      }
    }
  }

  std::ofstream outFile(filename);
  if (outFile.is_open()) {
    for (int i = 0; i < size; ++i) {
      for (int j = 0; j < size; ++j) {
        double max_value = values[i][j][0];
        for (int theta = 1; theta < theta_size; ++theta) {
          if (values[i][j][theta] > max_value) {
            max_value = values[i][j][theta];
          }
        }
        outFile << max_value << " ";
      }
      outFile << std::endl;
    }
    outFile.close();
  } else {
    std::cerr << "File cannot open" << std::endl;
  }
}

// CUDAメモリを開放する関数
void cleanup_cuda_memory() {
  hipFree(d_rewards);
  hipFree(d_values);
  hipFree(d_new_values);
  hipFree(d_actions);
}

int main() {
  int size = 200;           // マップサイズ設定
  int theta_size = 8;       // 各位置で進める方向の数
  double threshold = 1e-9;  // 収束判定閾値

  Matrix2D rewards;
  Matrix3D values;

  // 配列の初期化
  initialize_arrays(rewards, values, size, theta_size);

  // 各種設定
  set_goal(rewards, size);        // 目標位置の設定
  set_boundaries(rewards, size);  // 境界の設定
  set_puddle(rewards, size);      // 水たまりの設定
  set_obstacles(rewards, size);   // 障害物の設定

  // 目標位置の価値を初期化
  initialize_goal_values(values, size, theta_size);

  // アクションの生成
  std::vector<Action> actions = generate_actions();

  // CUDAメモリの初期化
  initialize_cuda_memory(rewards, values, actions, size, theta_size);

  // GPUの情報を表示
  print_gpu_info();

  // 計算時間の測定開始
  auto start = std::chrono::high_resolution_clock::now();

  // 値の反復計算を実行
  execute_value_iteration(size, theta_size, 1.0, 1000, threshold, actions);

  // 計算時間の測定終了
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  std::cout << "Elapsed time: " << elapsed.count() << " seconds" << std::endl;

  // 各グリッドにおいて最大の価値を計算して保存
  save_results("max_values.txt", size, theta_size);

  // CUDAメモリのクリーンアップ
  cleanup_cuda_memory();

  std::cout << "Value Iteration Complete !!!" << std::endl;
  return 0;
}
